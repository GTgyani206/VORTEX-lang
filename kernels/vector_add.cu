
#include <hip/hip_runtime.h>
extern "C" __global__ void add_vectors_kernel(const float* a, const float* b, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = a[idx] + b[idx];
    }
}
